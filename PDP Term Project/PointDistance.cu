#include "hip/hip_runtime.h"
//
//  PointDistance.cu
//  PDP Term Project
//
//  Created by Po-Chuan Wang on 2014/6/4
//  Copyright (c) 2014年 COLA Lab. All rights reserved.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Point.h"
#include "PointSet.h"
#include "PointDistance.h"

__device__ float norm(struct PointReal2D a, struct PointReal2D b)
{
    float ret = 0.0f;
    ret += (a.x - b.x) * (a.x - b.x);
    ret += (a.y - b.y) * (a.y - b.y);

    return sqrtf(ret);
}

__device__ float norm(struct PointReal3D a, struct PointReal3D b)
{
    float ret = 0.0f;
    ret += (a.x - b.x) * (a.x - b.x);
    ret += (a.y - b.y) * (a.y - b.y);
    ret += (a.z - b.z) * (a.z - b.z);

    return sqrtf(ret);
}

// Kernel for PointReal2D

// Kernel for PointReal3D

int BlockPointDistance(PointSet pointSet, float *deviceArray,
                       int hBegin, int hEnd, int vBegin, int vEnd)
{


    return 0;
}

